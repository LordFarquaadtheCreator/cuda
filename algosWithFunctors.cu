#include "hip/hip_runtime.h"
struct negate_float2
{
	__host__ __device__
	float2 operator()(float2 a)
	{
		return make_float2(-a.x, -a.y);
	}
};
// declare storage
device_vector<float2> input = ...
device_vector<float2> output = ...
// create functor
negate_float2 func;
// negate vectors
transform(input.begin(), input.end(), output.begin(), func);
